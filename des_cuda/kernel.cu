#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdint>
#include "des_cracker_cpu.cuh"
#include "des_cracker_gpu.cuh"
#include "runtime_parameters.hpp"

int main(int argc, char** argv)
{
	char *key_alphabet,
	     *plaintext_alphabet;
	uint64_t ciphertext;
	int key_length,
	    plaintext_length;
	bool run_cpu;

	parse_runtime_parameters(argc, argv, &key_alphabet, &key_length, &plaintext_alphabet, &plaintext_length, &ciphertext,
	                         &run_cpu);

	print_parameters(key_alphabet, key_length, plaintext_alphabet, plaintext_length, ciphertext, run_cpu);

	run_gpu_version(key_alphabet, key_length, plaintext_alphabet, plaintext_length, ciphertext, 100);
	if (run_cpu)
		run_cpu_version(key_alphabet, key_length, plaintext_alphabet, plaintext_length, ciphertext, 100);

	free(key_alphabet);
	free(plaintext_alphabet);

	return 0;
}
